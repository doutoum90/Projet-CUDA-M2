#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <ctime>
#include <time.h>
#include <sstream>
#include <string>
#include <fstream>

using namespace std;


__global__ void reduce0(int *g_idata, int *g_odata, int size){

	// paratagé avec tout les threads dans un bloc
	extern __shared__ int sdata[];

	//On recupere l'endroit ou l'on ecrit dans la memoire partagé ainsi que la case que nous allons traiter
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = 0;
	if (i<size)
		sdata[tid] = g_idata[i];
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce1(int *g_idata, int *g_odata, int size){

	// paratagé avec tout les threads dans un bloc
	extern __shared__ int sdata[];

	//On recupere l'endroit ou l'on ecrit dans la memoire partagé ainsi que la case que nous allons traiter
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = 0;
	if (i<size)
		sdata[tid] = g_idata[i];
	__syncthreads();

	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int index = 2 * s * tid;

		if (index < blockDim.x){
			sdata[index] += sdata[index + s];
		}
		__syncthreads();
	}

	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2(int *g_idata, int *g_odata, int size){

	// paratagé avec tout les threads dans un bloc
	extern __shared__ int sdata[];

	//On recupere l'endroit ou l'on ecrit dans la memoire partagé ainsi que la case que nous allons traiter
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = 0;
	if (i<size)
		sdata[tid] = g_idata[i];
	__syncthreads();

	for (unsigned int s =blockDim.x / 2; s>0; s >>= 1) {
		if(tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce3(int *g_idata, int *g_odata, int size){

	// paratagé avec tout les threads dans un bloc
	extern __shared__ int sdata[];

	//On recupere l'endroit ou l'on ecrit dans la memoire partagé ainsi que la case que nous allons traiter
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*( blockDim.x * 2) + threadIdx.x;
	if (i + blockDim.x <size)
		sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];

	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}

	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}


void benchmark(int size, void(*fptr)(int*, int*, int)){
	//initialize the data, all values will be 1(
	//so the final sum will be equal to size

	int threadsPerBlock = 1024;
	int totalBlocks = (size + (threadsPerBlock - 1)) / threadsPerBlock;


	cout << totalBlocks << " blocks, " << threadsPerBlock << " threads per blocks" << endl;


	//Without Optimization
	thrust::host_vector<int> data_h_i(size, 1);
	thrust::device_vector<int> data_v_i = data_h_i;
	thrust::device_vector<int> data_v_o(totalBlocks);

	int* output = thrust::raw_pointer_cast(data_v_o.data());
	int* input = thrust::raw_pointer_cast(data_v_i.data());

	hipEvent_t start, stop;
	float elapsedTime;

	hipEventCreate(&start);
	hipEventRecord(start, 0);


	fptr <<<totalBlocks, threadsPerBlock, threadsPerBlock*sizeof(int) >>>(input, output, size);

	//After the first call, now data_v_o contains the sum for each blocks so we need to sum them
	fptr <<<1, threadsPerBlock, threadsPerBlock*sizeof(int) >>>(output, input, totalBlocks);

	
	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	

	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed time : %f ms\n", elapsedTime);

	hipDeviceSynchronize();
	data_v_o[0] = data_v_i[0];
	data_v_i.clear();
	data_v_i.shrink_to_fit();

	thrust::host_vector<int> data_h_o = data_v_o;

	data_v_o.clear();
	data_v_o.shrink_to_fit();

	cout << "Somme : " << data_h_o[0] << endl;
}

void cpu_benchmark(int size){
	int sum = 0;
	thrust::host_vector<int> data_h_i(size, 1);

	clock_t cpu_startTime, cpu_endTime;

	double cpu_ElapseTime = 0;
	cpu_startTime = clock();
		
	for (int i = 0; i < size; i++){
		sum += data_h_i[i];
	}

	cpu_endTime = clock();

	cpu_ElapseTime = ((cpu_endTime - cpu_startTime));

	printf("Elapsed time : %f ms\n", cpu_ElapseTime);
	
	cout << "Somme : " << sum << endl;
}


int main(void){



	int size = 1048576;

	cout << "Size : " << size << endl;

	cout << "--------------------" << endl;
	cout << "CPU : " << endl;
	cpu_benchmark(size);

	cout << "--------------------" << endl;
	cout << "Without Optimization : " << endl;
	benchmark(size,&reduce0);
	
	cout << "--------------------" << endl;
	cout << "Without divergent branching : " << endl;
	benchmark(size, &reduce1);
	
	cout << "--------------------" << endl;
	cout << "With Sequential Adressing : " << endl;
	benchmark(size, &reduce2);

	cout << "--------------------" << endl;
	cout << "With No Idle Threads : " << endl;
	benchmark(size, &reduce3);


	return 0;
}
